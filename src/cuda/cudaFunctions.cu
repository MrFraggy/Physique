#include "cuda/cudaFunctions.hpp"

#include "leapFrog.ker"
#include "constantForces.ker"
#include "springBreak.ker"
#include "autoCollide.ker"

#include <iostream>
#include <ctime>

unsigned int d_MSize = 0;
// Masses
bool* d_Fixd = NULL;
glm::vec3* d_Pos = NULL;
glm::vec3* d_Vel = NULL;
glm::vec3* d_Frc = NULL;
float* d_Mass = NULL;

// Constant forces
unsigned int d_CstSize = 0;
glm::vec3* d_CstFrc = NULL;
glm::vec3* d_DevFrc = NULL;

// Spring breaks
int* d_MIds = NULL;
float* d_BLengths = NULL;
unsigned int d_MCount = 0;

void cudaLeapFrog(std::vector<unsigned char>& fxd, std::vector<glm::vec3>& p, 
	std::vector<glm::vec3>& v, std::vector<glm::vec3>& f, std::vector<float> mass) 
{
	
	if(d_Fixd == NULL)
	{
		d_MSize = fxd.size();

		hipMalloc((void**) &d_Fixd, sizeof(unsigned char)*d_MSize);
		hipMemcpy(d_Fixd, fxd.data(), sizeof(unsigned char)*d_MSize, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Pos, sizeof(glm::vec3)*d_MSize);
		hipMemcpy(d_Pos, p.data(), sizeof(glm::vec3)*d_MSize, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Vel, sizeof(glm::vec3)*d_MSize);
		hipMemcpy(d_Vel, v.data(), sizeof(glm::vec3)*d_MSize, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Frc, sizeof(glm::vec3)*d_MSize);
		hipMemcpy(d_Frc, f.data(), sizeof(glm::vec3)*d_MSize, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Mass, sizeof(float)*d_MSize);
		hipMemcpy(d_Mass, mass.data(), sizeof(float)*d_MSize, hipMemcpyHostToDevice);
	}

	dim3  grid( 5, 5, 1);
	dim3  threads( 5, 5, 1);

	// execute the kernel
	leapFrogKernel<<< grid, threads>>>(d_Fixd, d_Pos, d_Vel, d_Frc, d_Mass, d_MSize);
	
	/*hipMemcpy(fxd.data(), d_Fixd, sizeof(unsigned char)*d_MSize, hipMemcpyDeviceToHost);
	hipMemcpy(p.data(), d_Pos, sizeof(glm::vec3)*d_MSize, hipMemcpyDeviceToHost);
	hipMemcpy(v.data(), d_Vel, sizeof(glm::vec3)*d_MSize, hipMemcpyDeviceToHost);
	hipMemcpy(f.data(), d_Frc, sizeof(glm::vec3)*d_MSize, hipMemcpyDeviceToHost);
	hipMemcpy(mass.data(), d_Mass, sizeof(float)*d_MSize, hipMemcpyDeviceToHost);*/
}

void cudaConstantForces(std::vector<glm::vec3>& cf, std::vector<glm::vec3>& df)
{
	if(d_Pos == NULL)
	{
		return;
	}

	if(d_CstFrc == NULL)
	{
		d_CstSize = cf.size();
		hipMalloc((void**) &d_CstFrc, sizeof(glm::vec3)*d_CstSize);
		hipMemcpy(d_CstFrc, cf.data(), sizeof(glm::vec3)*d_CstSize, hipMemcpyHostToDevice);
		hipMalloc((void**) &d_DevFrc, sizeof(glm::vec3)*d_CstSize);
		hipMemcpy(d_DevFrc, df.data(), sizeof(glm::vec3)*d_CstSize, hipMemcpyHostToDevice);
	}
	glm::vec3 intensity = glm::linearRand(glm::vec3(-1,-1,-1),glm::vec3(1,1,1));
	dim3  grid( 5, 5, 1);
	dim3  threads( 5, 5, 1);

	// execute the kernel
	constantForcesKernel<<< grid, threads>>>(d_Frc, d_CstFrc, d_DevFrc, d_MSize, d_CstSize, intensity);
}

void cudaSpringbreak(std::vector<int>& mIds, std::vector<float>& blengths)
{
	if(d_Pos == NULL)
		return;

	if(d_MIds == NULL)
	{
		d_MCount = mIds.size();
		hipMalloc((void**) &d_MIds, sizeof(int)*d_MCount);
		hipMemcpy(d_MIds, mIds.data(), sizeof(int)*d_MCount, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_BLengths, sizeof(float)*d_MCount);
		hipMemcpy(d_BLengths, blengths.data(), sizeof(float)*d_MCount, hipMemcpyHostToDevice);
	}
	dim3  grid( 5*10, 5*10, 1);
	dim3  threads( 5, 5, 1);
	springbreakKernel<<< grid, threads>>>(d_Pos, d_Frc, d_Vel, d_MIds, d_BLengths, d_MCount/2);

	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		std::cerr << err << " " << hipGetErrorString(err) << std::endl;
}

void cudaAutoCollide()
{
	if(d_Pos == NULL)
		return;

	dim3  grid( 5*25, 5*25, 1);
	dim3  threads( 5, 5, 1);
	autoCollideKernel<<< grid, threads>>>(d_Pos, d_Frc, d_Vel, d_MCount);
}


bool cudaSnapshot(std::vector<glm::vec3>& pos)
{
	if(d_Pos)
	{
		hipMemcpy(pos.data(), d_Pos, sizeof(glm::vec3)*d_MSize, hipMemcpyDeviceToHost);
		return true;
	}
	return false;
}