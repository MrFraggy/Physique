#include "cuda/cudaFunctions.hpp"

#include "leapFrogKernel.icu"

bool* d_Fixd = NULL;
glm::vec3* d_Pos = NULL;
glm::vec3* d_Vel = NULL;
glm::vec3* d_Frc = NULL;
float* d_Mass = NULL;

void leapFrog(std::vector<bool>& fxd, std::vector<glm::vec3>& p, 
	std::vector<glm::vec3>& v, std::vector<glm::vec3>& f, std::vector<float> mass) 
{
	int size = fxd.size();
	if(d_Fixd == NULL)
	{
		//hipMalloc((void**) &d_Fixd, sizeof(bool)*size);
		//hipMemcpy(d_Fixd, fxd.data(), sizeof(bool)*size, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Pos, sizeof(glm::vec3)*size);
		hipMemcpy(d_Pos, p.data(), sizeof(glm::vec3)*size, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Vel, sizeof(glm::vec3)*size);
		hipMemcpy(d_Vel, v.data(), sizeof(glm::vec3)*size, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Frc, sizeof(glm::vec3)*size);
		hipMemcpy(d_Frc, f.data(), sizeof(glm::vec3)*size, hipMemcpyHostToDevice);

		hipMalloc((void**) &d_Mass, sizeof(float)*size);
		hipMemcpy(d_Mass, mass.data(), sizeof(float)*size, hipMemcpyHostToDevice);
	}

	dim3  grid( 1, 1, 1);
	dim3  threads( 128, 1, 1);

	// execute the kernel
	leapFrogKernel<<< grid, threads>>>(d_Fixd, d_Pos, d_Vel, d_Frc, d_Mass, size);
	//hipMemcpy(fxd.data(), d_Fixd, sizeof(bool)*size, hipMemcpyDeviceToHost);
	hipMemcpy(p.data(), d_Pos, sizeof(glm::vec3)*size, hipMemcpyDeviceToHost);
	hipMemcpy(v.data(), d_Vel, sizeof(glm::vec3)*size, hipMemcpyDeviceToHost);
	hipMemcpy(f.data(), d_Frc, sizeof(glm::vec3)*size, hipMemcpyDeviceToHost);
	hipMemcpy(mass.data(), d_Mass, sizeof(float)*size, hipMemcpyDeviceToHost);
}